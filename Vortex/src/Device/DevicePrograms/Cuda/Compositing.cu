#include "hip/hip_runtime.h"
#include "../CudaKernels.h"
#include ""
#include "Device/CUDAChecks.h"
#include "Device/DevicePrograms/ToneMapper.h"
#include "Device/DevicePrograms/LaunchParams.h"
#include "Device/Wrappers/dWrapper.h"

namespace vtx
{
	__forceinline__ __device__ math::vec3f fireflyRemoval(const math::vec3f* inputBuffer, const int x, const int y, const int width, const int height, const int kernelSize, const NoiseType noiseType, const float threshold)
	{
		// kernelSize is assumed to be an odd number
		const int halfKernel = kernelSize / 2;

		math::vec3f centerValue;
		if (noiseType == LUMINANCE)
		{
			centerValue = inputBuffer[y * width + x];
		}

		math::vec3f sumValue = math::vec3f(0.0f);
		int   count = 0;

		for (int dx = -halfKernel; dx <= halfKernel; ++dx)
		{
			for (int dy = -halfKernel; dy <= halfKernel; ++dy)
			{
				const int nx = x + dx;
				const int ny = y + dy;

				// exclude the center pixel
				if ((dx != 0 || dy != 0) && nx >= 0 && nx < width && ny >= 0 && ny < height)
				{
					sumValue += inputBuffer[ny * width + nx];
					count++;
				}
			}
		}

		// Calculate average luminance in the kernel (excluding center)
		float avgLuminance = utl::luminance(sumValue / count);

		// If the center pixel's luminance is higher than average plus threshold
		// replace it with average, otherwise keep the original
		if (utl::luminance(centerValue) > avgLuminance*threshold)
		{
			return sumValue / count;
		}

		return inputBuffer[y * width + x];
	}


	__global__ void fireFlyPass(LaunchParams* launchParams, int kernelSize, NoiseType noiseType, float threshold)
	{
		const int x = threadIdx.x + blockIdx.x * blockDim.x;
		const int y = threadIdx.y + blockIdx.y * blockDim.y;

		FrameBufferData* frameBuffer = &launchParams->frameBuffer;
		const math::vec2ui& frameSize = frameBuffer->frameSize;
		if (x >= frameSize.x || y >= frameSize.y) return;

		const uint32_t fbIndex = x + y * frameSize.x;

		/*const math::vec3f& directLightBuffer = frameBuffer->directLight[fbIndex];
		const math::vec3f& transmissionIndirect = frameBuffer->transmissionIndirect[fbIndex];

		const math::vec3f diffuse = fireflyRemoval(frameBuffer->diffuseIndirect, x, y, frameSize.x, frameSize.y, kernelSize, noiseType, threshold);
		const math::vec3f glossy = fireflyRemoval(frameBuffer->glossyIndirect, x, y, frameSize.x, frameSize.y, kernelSize, noiseType, threshold);
		const math::vec3f transmission = fireflyRemoval(frameBuffer->transmissionIndirect, x, y, frameSize.x, frameSize.y, kernelSize, noiseType, threshold);*/

		const math::vec3f filteredRadiance = fireflyRemoval(frameBuffer->hdriRadiance, x, y, frameSize.x, frameSize.y, kernelSize, noiseType, threshold);

		frameBuffer->fireflyPass[fbIndex] = filteredRadiance;
	}

	__forceinline__ __device__ void prepareOutput(math::vec3f* inputBuffer, LaunchParams* params, int pixelId, bool normalizeBySamples, bool dotoneMap)
	{
		math::vec3f output3f = inputBuffer[pixelId];
		if(normalizeBySamples)
		{
			output3f /= params->frameBuffer.samples[pixelId];
		}
		if(dotoneMap)
		{
			output3f = toneMap(params->toneMapperSettings, output3f);
		}
		reinterpret_cast<math::vec4f*>(params->frameBuffer.outputBuffer)[pixelId] = math::vec4f(output3f, 1.0f);
	}

	__forceinline__ __device__ void toneMapRadiance(const int id, const LaunchParams* params)
	{
		params->frameBuffer.hdriRadiance[id] = params->frameBuffer.radianceAccumulator[id] / params->frameBuffer.samples[id];
		params->frameBuffer.normalNormalized[id] = params->frameBuffer.normalAccumulator[id] / params->frameBuffer.samples[id];
		params->frameBuffer.albedoNormalized[id] = params->frameBuffer.albedoAccumulator[id] / params->frameBuffer.samples[id];
		params->frameBuffer.tmRadiance[id]= toneMap(params->toneMapperSettings, params->frameBuffer.hdriRadiance[id]);
	}

	__global__ void outputSelector(LaunchParams* launchParams, math::vec3f* beauty) {

		const int x = threadIdx.x + blockIdx.x * blockDim.x;
		const int y = threadIdx.y + blockIdx.y * blockDim.y;
		const FrameBufferData* frameBuffer = &launchParams->frameBuffer;
		const math::vec2ui& frameSize = frameBuffer->frameSize;
		if (x >= frameSize.x || y >= frameSize.y) return;

		const uint32_t fbIndex = x + y * frameSize.x;
		const RendererDeviceSettings* settings     = launchParams->settings;
		const auto                    outputBuffer = reinterpret_cast<math::vec4f*>(frameBuffer->outputBuffer);

		math::vec3f* input = nullptr;
		bool normalizeBySamples = true;
		bool dotoneMap = true;
		switch (settings->displayBuffer)
		{

		case(RendererDeviceSettings::DisplayBuffer::FB_BEAUTY):
		{
				if(beauty!= nullptr)
				{
					input = beauty;
					dotoneMap = true;
				}
				else
				{
					input = frameBuffer->tmRadiance;
					dotoneMap = false;
				}
				normalizeBySamples = false;
		}
		break;
		case(RendererDeviceSettings::DisplayBuffer::FB_NOISY):
		{
			input = frameBuffer->tmRadiance;
			dotoneMap = false;
			normalizeBySamples = false;
		}
		break;

		case(RendererDeviceSettings::DisplayBuffer::FB_DIFFUSE):
		{
				input = frameBuffer->albedoNormalized;
				dotoneMap = false;
				normalizeBySamples = false;
		}
		break;
		case(RendererDeviceSettings::DisplayBuffer::FB_ORIENTATION):
		{
				input = frameBuffer->orientation;
				dotoneMap = false;
		}
		break;
		case(RendererDeviceSettings::DisplayBuffer::FB_TRUE_NORMAL):
		{
				input = frameBuffer->trueNormal;
				dotoneMap = false;
		}
		break;
		case(RendererDeviceSettings::DisplayBuffer::FB_SHADING_NORMAL):
		{
				input = frameBuffer->normalNormalized;
				dotoneMap = false;
				normalizeBySamples = false;
		}
		break;
		case(RendererDeviceSettings::DisplayBuffer::FB_TANGENT):
		{
				input = frameBuffer->tangent;
				dotoneMap = false;
		}
		break;
		case(RendererDeviceSettings::DisplayBuffer::FB_UV):
		{
				input = frameBuffer->uv;
				dotoneMap = false;
		}
		break;
		case(RendererDeviceSettings::DisplayBuffer::FB_NOISE):
		{
			math:: vec3f value = floatToScientificRGB(frameBuffer->noiseBuffer[fbIndex].noiseAbsolute);
			outputBuffer[fbIndex] = math::vec4f(value, 1.0f);
		}
		break;
		case(RendererDeviceSettings::DisplayBuffer::FB_SAMPLES):
		{
			//const int maxPossibleSample = (launchParams->frameBuffer.frameSize.x * launchParams->frameBuffer.frameSize.y) * (launchParams->settings->iteration - launchParams->settings->minAdaptiveSamples) + launchParams->settings->iteration;
			//const int samplesDeltaRange = maxPossibleSample - launchParams->settings->minAdaptiveSamples;
			//const int samplesDelta = frameBuffer->samples[fbIndex] - launchParams->settings->minAdaptiveSamples;
			//float sampleMetric = (float)samplesDelta / (float)samplesDeltaRange;
			float sampleMetric = (float)(frameBuffer->samples[fbIndex] - launchParams->settings->minAdaptiveSamples)/ (float)(launchParams->settings->iteration- launchParams->settings->minAdaptiveSamples);
			sampleMetric *= 0.01f;
			sampleMetric = toneMap(launchParams->toneMapperSettings, math::vec3f(sampleMetric)).x;
			//sampleMetric = toneMap(launchParams->toneMapperSettings, math::vec3f(launchParams->frameBuffer.noiseBuffer[fbIndex].adaptiveSamples)).x;

			math::vec3f value = floatToScientificRGB(sampleMetric);
			//math::vec3f value = floatToScientificRGB(ACESFitted(frameBuffer->noiseBuffer[fbIndex].adaptiveSamples).x);
			outputBuffer[fbIndex] = math::vec4f(value, 1.0f);
		}
		break;
		case(RendererDeviceSettings::DisplayBuffer::FB_DEBUG_1):
		{
			outputBuffer[fbIndex] = math::vec4f(frameBuffer->debugColor1[fbIndex], 1.0f);
			dotoneMap = false;
		}
		break;
		}

		if(input!=nullptr)
		{
			prepareOutput(input, launchParams, fbIndex, normalizeBySamples, dotoneMap);
		}
	}

	void removeFireflies(LaunchParams* launchParams, int kernelSize, float threshold, int width, int height)
	{
		dim3 threadsPerBlock(16, 16);  // a common choice for 2D data
		// Calculate the number of blocks needed in each dimension
		dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

		fireFlyPass << <numBlocks, threadsPerBlock >> > (launchParams, kernelSize, LUMINANCE, threshold);
		//CUDA_CHECK(hipDeviceSynchronize());
	}

	void switchOutput(LaunchParams* launchParams, int width, int height, math::vec3f* beauty)
	{
		dim3 threadsPerBlock(16, 16);  // a common choice for 2D data
		// Calculate the number of blocks needed in each dimension
		dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

		outputSelector <<<numBlocks, threadsPerBlock>>>(launchParams, beauty);
		//CUDA_CHECK(hipDeviceSynchronize());
	}

	void toneMapRadianceKernel(const LaunchParams* launchParams, const int width, const int height, const char* name)
	{
		gpuParallelFor(name,
			width * height,
			[=] __device__(const int id)
		{
			toneMapRadiance(id, launchParams);
		});
	}
}
