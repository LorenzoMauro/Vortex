#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "Device/DevicePrograms/rendererFunctions.h"

namespace vtx
{

    extern "C" __global__ void wfShadeEntry(LaunchParams * params)
    {
        const unsigned int queueWorkId = blockIdx.x * blockDim.x + threadIdx.x;

        handleShading(queueWorkId, *params);
    }
}
